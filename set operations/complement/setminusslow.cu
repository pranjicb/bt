
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>

__global__ void setminus(int n, bool *x, bool *y){
    for(int i = 0; i < n; i++){
        x[i] = x[i] & !y[i];
    }
}


int main(void){

    int N = 1<<20;
    bool *x, *y, *z;

    hipMallocManaged(&x, N*sizeof(bool));
    hipMallocManaged(&y, N*sizeof(bool));
    hipMallocManaged(&z, N*sizeof(bool));

    for(int i = 0; i < N; i++){
        x[i] = rand() % 2;
        y[i] = rand() % 2;
        z[i] = x[i] & !y[i];
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    setminus<<<1,1>>>(N,x,y);
    hipDeviceSynchronize();

    int count = 0;
    for(int i = 0; i < N; i++){
        count += x[i] ^ z[i];
    }

    std::cout << "Error: " << count << std::endl;

    hipFree(x);
    hipFree(y);
    hipFree(z);

}