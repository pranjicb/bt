
#include <hip/hip_runtime.h>
#include "assert.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <fstream>

#define ll unsigned long long int

///////////////////////INPUT FILES//////////////////////////////////////
//BIO-SC-GT
std::string bioscgt_path = "../../../datasets/biological/bio-SC-GT/bio-SC-GT.edges";
int N_BIOSCGT = 1716; 
int E_BIOSCGT = 33987; 

//BIO-HUMAN-GENE2
std::string biohumangene2_path = "../../../datasets/biological/bio-human-gene2/bio-human-gene2.edges";
int N_BIOHUMANGENE2 = 14340; 
int E_BIOHUMANGENE2 = 9041364; 

//C500-9
std::string c500_path = "../../../datasets/dimac/C500-9/C500-9.mtx";
int N_C500 = 500;
int E_C500 = 112332;

//SC-PWTK
std::string scpwtk_path = "../../../datasets/scientific/sc-pwtk/sc-pwtk.mtx";
int N_SCPWTK = 217891;
int E_SCPWTK = 5653221;
/////////////////////////////////////////////////////////////////////////

void tccpu(int n, ll *sum, std::vector<std::vector<int>> g){
    #pragma omp parallel num_threads(32)
    {   
        #pragma omp for
        for(int i = 0; i < n; i++){
            std::vector<int> u = g.at(i);
            for(int j : u){
                if(i < j){
                    std::vector<int> v = g.at(j);
                    std::vector<int> w;
                    std::set_intersection(u.begin(), u.end(), v.begin(), v.end(), back_inserter(w));
                    sum[i] += w.size();
                }
                
            }
        } 
    }

}

//CUDA kernel for triangle counting using dense bitvectors
__global__ void tcdbgpu(ll n, ll *sum, bool *g){
    ll u = blockIdx.x * blockDim.x + threadIdx.x; //each thread calculates neighbor intersection for one node
    if(u < n){
        bool *u1 = &g[n*u];
        for(ll v = 0; v < u; v++){ //for all vertices in g
            if(g[n*u+v]){           //if v is a neighbor of u
                bool *v1 = &g[n*v];
                for(ll i = 0; i < n-7; i+=8){
                    ll tmp1,tmp2;
                    memcpy(&tmp1, &u1[i], 8);
                    memcpy(&tmp2, &v1[i], 8);
                    sum[u] += __popcll(tmp1&tmp2);
                }
            }
        }
    }
    
}


ll N = N_BIOSCGT;
ll E = E_BIOSCGT;
std::ifstream INPUT(bioscgt_path);

int main(){
    bool *g;
    hipMallocManaged(&g, N*N);
    //read input
    for(ll i = 0; i < E; i++){
        ll u,v;
        std::string w;
        INPUT >> u >> v >> w;
        g[N*u+v] = true;
    }
    //make graph undirected
    for(ll i = 0; i < N; i++){
        for(ll j = 0; j < N; j++){
            g[N*i+j] = g[N*i+j] | g[N*j+i];
        }
    }
    //remove self cycles
    for(ll i = 0; i < N; i++){
        g[N*i+i] = 0;
    }
    //make sparse array for CPU
    std::vector<std::vector<int>> gsa;
    for(ll i = 0; i < N; i++){
        std::vector<int> u;
        for(ll j = 0; j < N; j++){
            if(g[N*i+j]) u.push_back(j);
        }
        gsa.push_back(u);
    }

    //PARALLELIZED CPU IMPLEMENTATION
    ll *sumcpu;
    hipMallocManaged(&sumcpu, N*sizeof(ll));
    auto cpustart = std::chrono::steady_clock::now();
    tccpu(N,sumcpu, gsa);
    ll rescpu = 0;
    for(ll i = 0; i < N; i++){
        rescpu += sumcpu[i];
    }
    rescpu /= 3;
    auto cpuend = std::chrono::steady_clock::now();
    std::chrono::duration<double> cputime = cpuend-cpustart;


    //GPU IMPLEMENTATION BASED ON DENSE BITVECTORS
    ll *sumgpudb;
    hipMallocManaged(&sumgpudb, N*sizeof(ll));
    hipEvent_t dbstart, dbstop;
    hipEventCreate(&dbstart);
    hipEventCreate(&dbstop);
    
    int threadsPerBlock = 64;
    int blocksPerGrid = (N + threadsPerBlock - 1)/ threadsPerBlock;

    //launch and time kernel
    hipEventRecord(dbstart);
    tcdbgpu<<<blocksPerGrid, threadsPerBlock>>>(N, sumgpudb, g); 
    hipEventRecord(dbstop);

    hipDeviceSynchronize();
    hipEventSynchronize(dbstop);
    ll resgpudb = 0;
    for(ll i = 0; i < N; i++){
        resgpudb += sumgpudb[i];
    }
    resgpudb /= 3;
    float dbkerneltime = 0;
    hipEventElapsedTime(&dbkerneltime, dbstart, dbstop);

    std::cout << "TRUE RES: " << rescpu << std::endl;
    std::cout << "GPU RES: " << resgpudb << std::endl;
    std::cout << "CPU TIME: " << cputime.count() << "s" << std::endl;
    std::cout << "GPU TIME: " << dbkerneltime/1000.0 << "s" << std::endl;
    hipFree(g);
    hipFree(sumcpu);
    hipFree(sumgpudb);
}