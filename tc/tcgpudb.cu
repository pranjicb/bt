
#include <hip/hip_runtime.h>
#include "assert.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <fstream>
#include <string>

#define T unsigned long long int

///////////////////////INPUT FILES//////////////////////////////////////
//BIO-SC-GT
std::string bioscgt_path = "../../../datasets/biological/bio-SC-GT/bio-SC-GT.edges";
int N_BIOSCGT = 1716; 
int E_BIOSCGT = 33987; 

//BIO-HUMAN-GENE2
std::string biohumangene2_path = "../../../datasets/biological/bio-human-gene2/bio-human-gene2.edges";
int N_BIOHUMANGENE2 = 14340; 
int E_BIOHUMANGENE2 = 9041364; 

//C500-9
std::string c500_path = "../../../datasets/dimac/C500-9/C500-9.mtx";
int N_C500 = 500;
int E_C500 = 112332;

//SC-PWTK
std::string scpwtk_path = "../../../datasets/scientific/sc-pwtk/sc-pwtk.mtx";
int N_SCPWTK = 217891;
int E_SCPWTK = 5653221;
/////////////////////////////////////////////////////////////////////////

void tccpu(int n, T *sum, std::vector<std::vector<int>> g){
    #pragma omp parallel num_threads(32)
    {   
        #pragma omp for
        for(int i = 0; i < n; i++){
            std::vector<int> u = g.at(i);
            for(int j : u){
                if(i < j){
                    std::vector<int> v = g.at(j);
                    std::vector<int> w;
                    std::set_intersection(u.begin(), u.end(), v.begin(), v.end(), back_inserter(w));
                    sum[i] += w.size();
                }
                
            }
        } 
    }

}

//CUDA kernel for triangle counting using dense bitvectors
__global__ void tcdbgpu(int n, T *sum, bool *g){
    int u = blockIdx.x * blockDim.x + threadIdx.x; //each thread calculates neighbor intersection for one node
    if(u < n){
        bool *u1 = &g[n*u];
        for(int v = 0; v < u; v++){ //for all vertices in g
            if(g[n*u+v]){           //if v is a neighbor of u
                bool *v1 = &g[n*v];
                for(int i = 0; i < n-7; i+=8){
                    T tmp1,tmp2;
                    memcpy(&tmp1, &u1[i], 8);
                    memcpy(&tmp2, &v1[i], 8);
                    sum[u] += __popcll(tmp1&tmp2);
                }
            }
        }
    }
}

//CUDA kernel for triangle counting using dense bitvectors
__global__ void tcdbgpu2(int n1, int n2, T *sum, T *g){
    for(int i = 0; i < n1; i++){
        int count = 0;
        for(int j = 0; j < n2; j++){
            T tmp1 = g[n2*i+j];
            for(int k = 0; k < 64; k++){
                if(tmp1&(1>>k)!=0){
                    int v = 64*j+k;
                    for(int l = 0; l < n2; l++){
                        T tmp2 = g[n2*i+l];
                        //T tmp3 = g[n2*v+l];
                        T tmp3 = 0;
                        count += __popcll(tmp2&tmp3);
                    }
                }
            }
        }
        sum[i] += count;
    }
}

int numbits = sizeof(T) * 8;
unsigned int N = N_BIOSCGT;
unsigned int E = E_BIOSCGT;
std::ifstream INPUT(bioscgt_path);

int main(){
    bool *g;
    hipMallocManaged(&g, N*N);
    //read input
    for(int i = 0; i < E; i++){
        unsigned int u,v;
        std::string w;
        INPUT >> u >> v >> w;
        g[N*u+v] = true;
    }
    //make graph undirected
    for(int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            g[N*i+j] = g[N*i+j] | g[N*j+i];
        }
    }
    //remove self cycles
    for(int i = 0; i < N; i++){
        g[N*i+i] = 0;
    }


    
    //make sparse array for CPU
    std::vector<std::vector<int>> gsa;
    for(int i = 0; i < N; i++){
        std::vector<int> u;
        for(int j = 0; j < N; j++){
            if(g[N*i+j]) u.push_back(j);
        }
        gsa.push_back(u);
    }

    //PARALLELIZED CPU IMPLEMENTATION
    T *sumcpu;
    hipMallocManaged(&sumcpu, N*sizeof(T));
    auto cpustart = std::chrono::steady_clock::now();
    tccpu(N,sumcpu, gsa);
    T rescpu = 0;
    for(int i = 0; i < N; i++){
        rescpu += sumcpu[i];
    }
    rescpu /= 3;
    auto cpuend = std::chrono::steady_clock::now();
    std::chrono::duration<double> cputime = cpuend-cpustart;


    //GPU IMPLEMENTATION BASED ON DENSE BITVECTORS
    T *sumgpudb;
    hipMallocManaged(&sumgpudb, N*sizeof(T));
    hipEvent_t dbstart, dbstop;
    hipEventCreate(&dbstart);
    hipEventCreate(&dbstop);
    
    int threadsPerBlock = 64;
    int blocksPerGrid = (N + threadsPerBlock - 1)/ threadsPerBlock;

    //launch and time kernel
    hipEventRecord(dbstart);
    tcdbgpu<<<blocksPerGrid, threadsPerBlock>>>(N, sumgpudb, g); 
    hipEventRecord(dbstop);

    hipDeviceSynchronize();
    hipEventSynchronize(dbstop);
    T resgpudb = 0;
    for(int i = 0; i < N; i++){
        resgpudb += sumgpudb[i];
    }
    resgpudb /= 3;
    float dbkerneltime = 0;
    hipEventElapsedTime(&dbkerneltime, dbstart, dbstop);

    

    std::cout << "TRUE RES: " << rescpu << std::endl;
    std::cout << "GPU RES: " << resgpudb << std::endl;
    std::cout << "CPU TIME: " << cputime.count() << "s" << std::endl;
    std::cout << "GPU TIME: " << dbkerneltime/1000.0 << "s" << std::endl;
    std::cout << N << " " << E << std::endl;


    hipFree(g);
    hipFree(sumcpu);
    hipFree(sumgpudb);
}