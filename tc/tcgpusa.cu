
#include <hip/hip_runtime.h>
#include "assert.h"
#include <iostream>
#include <chrono>
#include <vector>
#include <algorithm>
#include <fstream>

#define T unsigned long long int

///////////////////////INPUT FILES//////////////////////////////////////
//BIO-SC-GT
std::string bioscgt_path = "../../../datasets/biological/bio-SC-GT/bio-SC-GT.edges";
int N_BIOSCGT = 1716; 
int E_BIOSCGT = 33987; 

//BIO-HUMAN-GENE2
std::string biohumangene2_path = "../../../datasets/biological/bio-human-gene2/bio-human-gene2.edges";
int N_BIOHUMANGENE2 = 14340; 
int E_BIOHUMANGENE2 = 9041364; 

//C500-9
std::string c500_path = "../../../datasets/dimac/C500-9/C500-9.mtx";
int N_C500 = 500;
int E_C500 = 112332;

//SC-PWTK
std::string scpwtk_path = "../../../datasets/scientific/sc-pwtk/sc-pwtk.mtx";
int N_SCPWTK = 217891;
int E_SCPWTK = 5653221;
/////////////////////////////////////////////////////////////////////////

__device__ T intersectCount(T lu, T ru, T lv, T rv, T *edges1, T *edges2, T *nodes){
    T count = 0;
    for(int i = lu; i < ru; i++){
        for(int j = lv; j < rv; j++){
            if(edges2[i] == edges2[j]) count++;
        }
    }
    return count;
}

__global__ void tcgpusa(T N, T E, T *nodes, T *edges1, T*edges2, T *sum){
    T t = blockIdx.x * blockDim.x + threadIdx.x;
    if(true){
        T u = edges1[t];
        T v = edges2[t];
        T counter = 0;
        if(u < v){
            T lu = nodes[u];
            T lv = nodes[v];
            T ru, rv;
            if(u != N-1){
                ru = nodes[u+1];
            }
            else{
                ru = E-1;
            }
            if(v != N-1){
                rv = nodes[v+1];
            }
            else{
                rv = E-1;
            }
            T sizeU = ru-lu;
            T sizeV = rv-lu;
            if(sizeU < sizeV){
                counter += intersectCount(lu,ru,lv,rv,edges1,edges2,nodes);
            }
            else{
                counter += intersectCount(lv,rv,lu,ru,edges1,edges2,nodes);
            }
            
        }
        sum[t] = counter;

    }
}

void tccpu(int n, T *sum, std::vector<std::vector<int>> g){
    #pragma omp parallel num_threads(32)
    {   
        #pragma omp for
        for(int i = 0; i < n; i++){
            std::vector<int> u = g.at(i);
            for(int j : u){
                if(i < j){
                    std::vector<int> v = g.at(j);
                    std::vector<int> w;
                    std::set_intersection(u.begin(), u.end(), v.begin(), v.end(), back_inserter(w));
                    sum[i] += w.size();
                }
                
            }
        } 
    }

}


//CPU implementation for triangle counting
int count_triangles(int n, bool *g){
    int count = 0; 
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            for(int k = 0; k < n; k++){
                if(g[n*i+j] && g[n*j+k] && g[n*k+i]) count++;
            }
        }
    }
    return count;
}

T N = N_BIOSCGT;
T E = E_BIOSCGT*2;
std::ifstream INPUT(bioscgt_path);

int main(){
    bool *g;
    hipMallocManaged(&g, N*N);
    
    //read input
    for(T i = 0; i < E; i++){
        T u,v;
        std::string w;
        INPUT >> u >> v >> w;
        g[N*u+v] = true;
    }
    //make graph undirected
    for(T i = 0; i < N; i++){
        for(T j = 0; j < N; j++){
            g[N*i+j] = g[N*i+j] | g[N*j+i];
        }
    }
    //remove self cycles
    for(T i = 0; i < N; i++){
        g[N*i+i] = 0;
    }
    

    //make sparse array for CPU
    std::vector<std::vector<int>> gsa;
    for(T i = 0; i < N; i++){
        std::vector<int> u;
        for(T j = 0; j < N; j++){
            if(g[N*i+j]) u.push_back(j);
        }
        gsa.push_back(u);
    }

    //PARATELIZED CPU IMPLEMENTATION
    T *sumcpu;
    hipMallocManaged(&sumcpu, N*sizeof(T));
    auto cpustart = std::chrono::steady_clock::now();
    tccpu(N,sumcpu, gsa);
    T rescpu = 0;
    for(T i = 0; i < N; i++){
        rescpu += sumcpu[i];
    }
    rescpu /= 3;
    auto cpuend = std::chrono::steady_clock::now();
    std::chrono::duration<double> cputime = cpuend-cpustart;


    //GPU SA IMPLEMENTATION

    T *edges1;
    hipMallocManaged(&edges1, E*sizeof(T));
    T *edges2;
    hipMallocManaged(&edges2, E*sizeof(T));
    T *nodes;
    hipMallocManaged(&nodes, N*sizeof(T));

    for(int i = 0; i < E; i++){
        edges1[i] = 0;
        edges2[i] = 0;
    }

    for(int i = 0; i < N; i++){
        nodes[i] = 0;
    }

    T idx = 0;
    for(T i = 0; i < N; i++){
        nodes[i] = idx;
        for(T j = 0; j < N; j++){
            if(g[N*i+j] && i!=j){
                edges1[idx] = i;
                edges2[idx] = j;
                idx++;
            }
        }
    }

    T *sumgpusa;
    hipMallocManaged(&sumgpusa, E*sizeof(T));
    hipEvent_t sastart, sastop;
    hipEventCreate(&sastart);
    hipEventCreate(&sastop);
    
    int threadsPerBlock = 64;
    int blocksPerGrid = (E + threadsPerBlock - 1)/ threadsPerBlock;

    //launch and time kernel
    hipEventRecord(sastart);
    tcgpusa<<<blocksPerGrid, threadsPerBlock>>>(N, E, nodes, edges1, edges2, sumgpusa); 
    hipEventRecord(sastop);

    hipDeviceSynchronize();
    hipEventSynchronize(sastop);
    T resgpusa = 0;
    for(T i = 0; i < E; i++){
        resgpusa += sumgpusa[i];
    }
    resgpusa /= 3;
    float sakerneltime = 0;
    hipEventElapsedTime(&sakerneltime, sastart, sastop);

    std::cout << "TRUE RES: " << rescpu << std::endl;
    std::cout << "GPU SA RES: " << resgpusa << std::endl;
    std::cout << "CPU TIME: " << cputime.count() << "s" << std::endl;
    std::cout << "GPU SA TIME: " << sakerneltime/1000.0 << "s" << std::endl;
    std::cout << "CPU slow res: " << count_triangles(N,g)/6 << std::endl; 

    hipFree(sumgpusa);
    hipFree(edges1);
    hipFree(edges2);
    hipFree(nodes);
    hipFree(g);
    hipFree(sumcpu);
}